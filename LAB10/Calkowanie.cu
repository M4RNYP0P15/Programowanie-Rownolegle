#include "hip/hip_runtime.h"
// #include <stdio.h>
// #include <stdlib.h>
#include <hip/hip_cooperative_groups.h>
#include <iostream>
using namespace std;
namespace cg = cooperative_groups;


#define H_ERR(a)\
{\
    hipError_t status = a;\
    if(status!= hipSuccess){ printf("%s", hipGetErrorString(status)); exit(0);}\
}

inline double funcc(double x){
    return 3*x*x-6*x+5;
}

inline __device__ double func(double x){
    return 3*x*x-6*x+5;
}

bool isPow2(unsigned int x) { return ((x & (x - 1)) == 0); }

template <unsigned int blockSize, bool nIsPow2>
__global__ void redukcja7(unsigned int n, double *g_idata, double *g_odata) {
  cg::thread_block cta = cg::this_thread_block();
  //double *sdata = SharedMemory<double>();
  extern __shared__ double sdata[];

  unsigned int tid = threadIdx.x;
  unsigned int gridSize = blockSize * gridDim.x;

  double mySum = 0;

  // we reduce multiple elements per thread.  The number is determined by the
  // number of active thread blocks (via gridDim).  More blocks will result in a larger gridSize and therefore fewer elements per thread
  if (nIsPow2) {
    unsigned int i = blockIdx.x * blockSize * 2 + threadIdx.x;
    gridSize = gridSize << 1;

    while (i < n) {
      mySum += g_idata[i];
      if ((i + blockSize) < n) {
        mySum += g_idata[i + blockSize];
      }
      i += gridSize;
    }
  } else {
    unsigned int i = blockIdx.x * blockSize + threadIdx.x;
    while (i < n) {
      mySum += g_idata[i];
      i += gridSize;
    }
  }

  sdata[tid] = mySum;
  cg::sync(cta);

  if ((blockSize >= 512) && (tid < 256)) {  sdata[tid] = mySum = mySum + sdata[tid + 256];  }

  cg::sync(cta);

  if ((blockSize >= 256) && (tid < 128)) {   sdata[tid] = mySum = mySum + sdata[tid + 128]; }

  cg::sync(cta);

  if ((blockSize >= 128) && (tid < 64)) { sdata[tid] = mySum = mySum + sdata[tid + 64]; }

  cg::sync(cta);

  cg::thread_block_tile<32> tile32 = cg::tiled_partition<32>(cta);

  if (cta.thread_rank() < 32) {
    // Fetch final intermediate sum from 2nd warp
    if (blockSize >= 64) mySum += sdata[tid + 32];
    for (int offset = tile32.size() / 2; offset > 0; offset /= 2) {
      mySum += tile32.shfl_down(mySum, offset);
    }
  }

  if (cta.thread_rank() == 0) g_odata[blockIdx.x] = mySum;
}

__global__ void M_Trapezow(double a, double b, double dx, int n, double * wyniki){
    unsigned int g_x = blockIdx.x*blockDim.x+ threadIdx.x;
    if(g_x == 0){
        wyniki[g_x] = dx*(func(a)+func(b))/2.0;
        //wyniki[g_x] =1;
    }
    if(g_x > 0 && g_x< n-1){
        wyniki[g_x] = func(a+g_x*dx)*dx;
        //wyniki[g_x] =1;
    }
}

__global__ void M_Prostokatow(double a, double b, double dx, int n, double * wyniki){
    unsigned int g_x = blockIdx.x*blockDim.x+ threadIdx.x;
    if(g_x == 0){
        wyniki[g_x] = 0;
    }
    if(g_x > 0 && g_x<= n){
        wyniki[g_x] = func(a + g_x*dx)*dx;
    }
}

__global__ void M_Simpson(double a, double b, double dx, int n, double * wyniki, double *swyn){
    unsigned int g_x = blockIdx.x*blockDim.x+ threadIdx.x;
    if(g_x == 0){
        swyn[g_x] = func(b - dx / 2);
    }
    if(g_x > 0 && g_x< n){
        wyniki[g_x] = func(a + g_x*dx);
        swyn[g_x] = func((a + g_x*dx) - dx/2);
    }
}

double Sumuj(int SIZE, double * inputDoSum){
    double* buffer1;
    double* buffer2;
    double* buf_in;
    double* buf_out;
    double* tmp_buf;
    double* buf;
    int Blocks_red = 128;
    double* result_host;
    unsigned int buf_size = (SIZE-1)/Blocks_red+1;
    H_ERR( hipMalloc ((void**)&buffer1 ,buf_size * sizeof(double))    );
    H_ERR( hipHostMalloc( (void**) &result_host , 1 * sizeof(double))  );
    H_ERR( hipMalloc ((void**)&buffer2 ,(buf_size/Blocks_red +1) * sizeof(double))    );
    H_ERR( hipMalloc ((void**)&buf ,1 * sizeof(double))    );
    // buf_size = (SIZE-1)/Blocks_red+1;
    //GridSize = (n-1)/Blocks_red +1; // Dobry wynik dla SIZE >1
    long int GridSize = ((SIZE+Blocks_red-1))/Blocks_red;
    long int MySize = SIZE;
    buf_in = inputDoSum;
    buf_out = buffer1;

    if (isPow2(GridSize)) {
        switch (Blocks_red){
            case 512:
            redukcja7<512, true><<<GridSize/2,Blocks_red,Blocks_red*sizeof(double) >>>(MySize, buf_in, buf_out); break;
            case 256:
            redukcja7<256, true><<<GridSize/2,Blocks_red,Blocks_red*sizeof(double) >>>(MySize, buf_in, buf_out); break;
            case 128:
            redukcja7<128, true><<<GridSize/2,Blocks_red,Blocks_red*sizeof(double) >>>(MySize, buf_in, buf_out); break;
        }
    }else
    {
        switch (Blocks_red){
            case 512:
            redukcja7<512, false><<<GridSize,Blocks_red,Blocks_red*sizeof(double) >>>(MySize, buf_in, buf_out); break;
            case 256:
            redukcja7<256, false><<<GridSize,Blocks_red,Blocks_red*sizeof(double) >>>(MySize, buf_in, buf_out); break;
            case 128:
            redukcja7<128, false><<<GridSize,Blocks_red,Blocks_red*sizeof(double) >>>(MySize, buf_in, buf_out); break;
        }
    }

    // Przygotowanie do wejścia do pętli
    buf_in = buffer1;
    buf_out = buffer2;
    MySize=GridSize;
    //GridSize = (GridSize-1)/Blocks_red +1; 
    GridSize = (GridSize+Blocks_red-1)/Blocks_red;
    while (GridSize >1 ) { 
        if (isPow2(GridSize)) {
            switch (Blocks_red){
                case 512:
                redukcja7<512, true><<<GridSize/2,Blocks_red,Blocks_red*sizeof(double) >>>(MySize, buf_in, buf_out); break;
                case 256:
                redukcja7<256, true><<<GridSize/2,Blocks_red,Blocks_red*sizeof(double) >>>(MySize, buf_in, buf_out); break;
                case 128:
                redukcja7<128, true><<<GridSize/2,Blocks_red,Blocks_red*sizeof(double) >>>(MySize, buf_in, buf_out); break;
                }
                break;
        }
        else{
            switch (Blocks_red){
                case 512:
                redukcja7<512, false><<<GridSize,Blocks_red,Blocks_red*sizeof(double) >>>(MySize, buf_in, buf_out); break;
                case 256:
                redukcja7<256, false><<<GridSize,Blocks_red,Blocks_red*sizeof(double) >>>(MySize, buf_in, buf_out); break;
                case 128:
                redukcja7<128, false><<<GridSize,Blocks_red,Blocks_red*sizeof(double) >>>(MySize, buf_in, buf_out); break;
                }
            break;
            }
        // Zamieniamy miejscami bufory robocze
        tmp_buf = buf_in;
        buf_in = buf_out;
        buf_out = tmp_buf;
        // 
        MySize=GridSize;
        //GridSize = (GridSize-1)/Blocks_red +1;
        GridSize = (GridSize+Blocks_red-1)/Blocks_red;
    }
    buf_out = buf;
    if (isPow2(GridSize)) {
        switch (Blocks_red){
            case 512:
                redukcja7<512, true><<<GridSize,Blocks_red,Blocks_red*sizeof(double) >>>(MySize, buf_in, buf_out); break;
            case 256:
                redukcja7<256, true><<<GridSize,Blocks_red,Blocks_red*sizeof(double) >>>(MySize, buf_in, buf_out); break;
            case 128:
                redukcja7<128, true><<<GridSize,Blocks_red,Blocks_red*sizeof(double) >>>(MySize, buf_in, buf_out); break;
        }
    }
    else
    {
        switch (Blocks_red)
        {
            case 512:
                redukcja7<512, false><<<GridSize,Blocks_red,Blocks_red*sizeof(double) >>>(MySize, buf_in, buf_out); break;
            case 256:
                redukcja7<256, false><<<GridSize,Blocks_red,Blocks_red*sizeof(double) >>>(MySize, buf_in, buf_out); break;
            case 128:
                redukcja7<128, false><<<GridSize,Blocks_red,Blocks_red*sizeof(double) >>>(MySize, buf_in, buf_out); break;
        }
    }
    H_ERR( hipMemcpy( result_host, buf, 1*sizeof(double), hipMemcpyDeviceToHost)    );
    //printf("%lf \n", result_host[0]);
    return result_host[0];
}

int main(int argc, char **argv)
{
    double xp,xk;
    long int n;
    int blocks = 1;
    int threads;
    //dim3 threadsPerBlock(128,1,1);
    //dim3 numBlocks((n)/threadsPerBlock.x+1,1,1);
    // printf("Podaj poczatek przedzialu calkowania\n");
    // scanf("%f", &xp);
    // printf("Podaj koniec przedzialu calkowania\n");
    // scanf("%f", &xk);
    float exec_time;

    xp=1.0;
    xk=2.0;
    printf("Przedzial xp: %lf, xk: %lf\n", xp, xk);
    // printf("Podaj dokladnosc calkowania\n");
    // scanf("%d", &n);

    n = atoi(argv[1]);
    printf("%ld\n", n);
    double dx = (xk - xp)/(float)n;
    double* wyniki_c;
    double* s_wyniki;
    double s;
    double calka;
    hipEvent_t event1, event2;
    hipEventCreate(&event1);
    hipEventCreate(&event2);
    
    H_ERR( hipMalloc ((void**)&wyniki_c, n*sizeof(double))    );
    H_ERR( hipMalloc ((void**)&s_wyniki, n*sizeof(double))    );
    if(n>1024){
        blocks = (n + 1023)/1024;
        threads = 1024;
    }else{
        threads = n;
    }
    //////////////// Trapezy ////////////
    hipEventRecord(event1, 0);
    M_Trapezow<<<blocks, threads, 0>>>(xp, xk, dx, n, wyniki_c);
    H_ERR(  hipDeviceSynchronize() ); 
    // wstawić sumowanie tablicy na GPU
    calka = Sumuj(n, wyniki_c);
    
    
    hipEventRecord(event2,0);
    hipEventSynchronize(event2);
    hipEventElapsedTime(&exec_time, event1, event2);
    printf("Wynik metoda trapezow: %f \t czas: %f \n", calka, exec_time);
    /////////////// Prostokaty ///////////////
    hipEventRecord(event1, 0);
    M_Prostokatow<<<blocks, threads, 0>>>(xp, xk, dx, n, wyniki_c);
    H_ERR(  hipDeviceSynchronize() ); 
    // wstawić sumowanie wynikic
    calka = Sumuj(n, wyniki_c);

    hipEventRecord(event2,0);
    hipEventSynchronize(event2);
    hipEventElapsedTime(&exec_time, event1, event2);
    printf("Wynik metoda prostokatow: %f  \t czas: %f\n", calka, exec_time);

    ////////// Simpson ////////////////

    hipEventRecord(event1, 0);
    M_Simpson<<<blocks, threads, 0>>>(xp, xk, dx, n, wyniki_c, s_wyniki);
    H_ERR(  hipDeviceSynchronize() ); 
    // wstawić sumowanie wynikic i swyniki
    calka = Sumuj(n, wyniki_c);
    s = Sumuj(n, s_wyniki);
    //H_ERR( hipMemcpy( s, buffer1, sizeof(double), hipMemcpyDeviceToHost)    );  // buffer1
    hipEventRecord(event2,0);
    hipEventSynchronize(event2);
    hipEventElapsedTime(&exec_time, event1, event2);
    printf("Wynik metoda simpsona: %f \t czas: %f\n", ( (dx/6) * (funcc(xp) + funcc(xk) + 2*calka + 4*s)), exec_time );
}
