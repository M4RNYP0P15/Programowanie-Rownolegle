#define NO_FREETYPE 


#include <hip/hip_runtime.h>
#include <math.h>
#include <chrono>
#include <iostream> 
using namespace std;
#include <sys/time.h>
#define real double

__global__ void redukcja1 (long long N, int* v, long long* out, int blockSize)  // N wielkość tablicy wejściowej; v- tablica wejsciowa(z której liczymy) ; out -...
{
 size_t s = threadIdx.x + blockIdx.x * blockDim.x; // indeks globalny wątku
 int sID = threadIdx.x;  // indeks lokalny wątku 
 size_t i;

 __shared__ long long pom[1024]; // deklatujemy tablice pomocniczną w pamięci współdzielonej
 
 if (s<N)  // sprawdzamy czy indeks globalny jest mniejszy od wielkosci tablicy wejsciowej
	pom[sID] = v[s]; // przypisanie do tablicy pomocnicznej o indeksie watku lokalnego elementu z tablicy wejciowej o indeksie watku globalnego
 else 
    pom[sID] = 0;

 __syncthreads(); // czekamy az wszystkie watki w bloku dotra do tego miejsca

 for (i=1; i<blockSize; i*=2){
    int ad = 2*i*sID;
 	if (ad<blockSize){
		pom[ad] += pom[ad + i];
 	}		
 	__syncthreads();
 }
 if (sID==0) out[blockIdx.x] = pom[0];  // na koniec przypisujemy w tablicy out ( o indeksach bloku) 
}

__global__ void redukcja (int N, long long* v, long long* out, int blockSize)  // N wielkość tablicy wejściowej; v- tablica wejsciowa(z której liczymy) ; out -...
{
 size_t s = threadIdx.x + blockIdx.x * blockDim.x; // indeks globalny wątku
 int sID = threadIdx.x;  // indeks lokalny wątku 
 size_t i;

 __shared__ long long pom[1024]; // deklatujemy tablice pomocniczną w pamięci współdzielonej
 
 if (s<N)  // sprawdzamy czy indeks globalny jest mniejszy od wielkosci tablicy wejsciowej
	pom[sID] = v[s]; // przypisanie do tablicy pomocnicznej o indeksie watku lokalnego elementu z tablicy wejciowej o indeksie watku globalnego
 else 
    pom[sID] = 0;

 __syncthreads(); // czekamy az wszystkie watki w bloku dotra do tego miejsca

 for (i=1; i<blockSize; i*=2){
    int ad = 2*i*sID;
 	if (ad<blockSize){
		pom[ad] += pom[ad + i];
 	}		
 	__syncthreads();
 }
 if (sID==0) out[blockIdx.x] = pom[0];  // na koniec przypisujemy w tablicy out ( o indeksach bloku) 
}

__global__ void cudaMandelbrot(real X0, real Y0, real X1, real Y1, int POZ, int PION, int ITER,int *Mandel ){
    int indx = blockIdx.x * blockDim.x + threadIdx.x;
    int k;
    int SIZE = POZ*PION;
    double X_re, X_im, Y_re, Y_im, d_re, d_im, t_re;

    d_re = (X1-X0)/(POZ-1);
    d_im = (Y1-Y0)/(PION-1);
    if(indx<SIZE){
        X_re= X0+ d_re*(indx%POZ);
        X_im= Y0+ (indx/POZ)*d_im;
        Y_re= X_re;
        Y_im= X_im;
        k=0;
        while(k<ITER & (Y_re*Y_re + Y_im*Y_im)<4){
           t_re = Y_re*Y_re-Y_im*Y_im+X_re;
           Y_im = 2*Y_re*Y_im+X_im;
           Y_re = t_re;
           k++;
        }
        Mandel[indx]= k;
    }
}

double srednia(int len, double *tab){
    double sum=0;
    for(int i =1;i<len; i++ ){
        sum+=tab[i]/(len-1);
    }
    return sum;
}

long long computeMandelbrot(real X0, real Y0, real X1, real Y1, int POZ, int PION, int ITER,int *Mandel ){
    int k;
    long long SUM=0;
    real X_re, X_im, Y_re, Y_im, d_re, d_im, t_re;

    d_re = (X1-X0)/(POZ-1);
    d_im = (Y1-Y0)/(PION-1);

    for(int i =0; i<PION; i++){
        for(int j =0; j<POZ; j++){
            X_re= X0+ d_re*j;
            X_im = Y0+ i*d_im;
            Y_re= X_re;
            Y_im= X_im;
            k=0;
            while(k<ITER & (Y_re*Y_re + Y_im*Y_im)<4){
                t_re = Y_re*Y_re-Y_im*Y_im+X_re;
                Y_im = 2*Y_re*Y_im+X_im;
                Y_re = t_re;
                k++;
            }
            Mandel[i*POZ+j]= k;
            SUM+=k;
        }
    }
    return SUM;
}

int main(int argc, char **argv) {
    //Ustaw obszar obliczeń {X0,Y0} - lewy dolny róg
    double X0=-1.0;    double Y0=0.0;
    //{X1,Y1} - prawy górny róg
	double X1=-0.5;    double Y1=0.5;
    //Ustal liczbę iteracji próbkowania {ITER}
    int ITER=256;

    int* Iters_gpu;
    int POZ, PION;

    int ile_iter=50;
    double wyniki[ile_iter];
    dim3 threadPerBlock(1024,1,1);
    hipError_t status;
    for(int ik =1; ik<=10; ik++){
        
        POZ=1000*ik; PION=1000*ik; //Ustal rozmiar w pikselach {POZ,PION}
        cout << "Dla wymiaru obrazka:"<< POZ << "x" << PION <<endl;
        int *Iters = (int*) malloc(sizeof(int)*POZ*PION); //Zaalokuj tablicę do przechowywania wyniku

        status = hipMalloc((void**)&Iters_gpu, sizeof(int)* POZ*PION);
        if(status!= hipSuccess){ cout << hipGetErrorString(status) << endl;}
        
        dim3 numBlocks(PION*POZ/threadPerBlock.x+1,1,1);
        for(int ij=0;ij<ile_iter;ij++)
        {
            auto start2 = chrono::steady_clock::now(); 
            cudaMandelbrot<<<numBlocks,threadPerBlock>>>(X0,Y0,X1,Y1,POZ,PION,ITER,Iters_gpu);
            hipMemcpy(Iters, Iters_gpu, sizeof(int)*POZ*PION, hipMemcpyDeviceToHost);
            auto stop = chrono::steady_clock::now();
            auto diff = stop - start2;
            
            // cout << chrono::duration <double, milli> (diff).count() << endl;
            wyniki[ij]=chrono::duration <double, milli> (diff).count();
        }
        cout << "Średnia: " << srednia(ile_iter, wyniki) << endl;
    }

    // int blockSize = 128;
    //   size_t N = POZ*PION;
    //   int blocks = (N + blockSize-1) / blockSize;
    //   int blocks1 = (blocks + blockSize-1) / blockSize;

    //     long long* outV;
    //     cudaMalloc( (void**) &outV, blocks * sizeof(long long) );
    //     long long* outV1;
    //     cudaMalloc( (void**) &outV1, blocks1 * sizeof(long long) );
    //     long long out;

    //         redukcja1 <<<blocks, blockSize>>>(POZ*PION,Iters_gpu, outV, blockSize);
    //         blocks1 = (blocks + blockSize-1) / blockSize;
    //         while(blocks1 > 0){
    //             //printf("blocks: %d  1: %d \n", blocks, blocks1);
    //             redukcja<<<blocks1, blockSize>>> (blocks, outV, outV1, blockSize);
    //             blocks = blocks1;
    //             blocks1 = (blocks1 + blockSize-1) / blockSize;
    //             redukcja<<<blocks1, blockSize>>> (blocks, outV1, outV, blockSize);
    //             if(blocks1 == 1) break;
    //             blocks = blocks1;
    //             blocks1 = (blocks1 + blockSize-1) / blockSize;
    //         }
    //     cudaDeviceSynchronize();

    //     cudaMemcpy (&out, outV, 1 * sizeof(long long), cudaMemcpyDeviceToHost);
    //     printf ("GPU wynik %lld; \n", out);

    // status = cudaFree(Iters_gpu);
    // if(status != cudaSuccess){ cout << cudaGetErrorString(status) << endl;}
    // int *Iters_cpu = (int*) malloc(sizeof(int)*POZ*PION);
    // auto start1 = chrono::steady_clock::now();
    // long long SUMA_CPU = computeMandelbrot(X0,Y0,X1,Y1,POZ,PION,ITER,Iters_cpu);
     
    // auto stop1 = chrono::steady_clock::now();
    // auto diff1 = stop1 - start1;
    // cout << "CPU czas :"<<chrono::duration <double, milli> (diff1).count() <<" Suma:"<< SUMA_CPU << endl;
    
   
    //free(Iters);
}
