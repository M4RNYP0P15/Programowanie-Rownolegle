#include "hip/hip_runtime.h"
%%cu
#define NO_FREETYPE 

#include <math.h>
#include <chrono>
#include <iostream> 
using namespace std;
#include <sys/time.h>
#define real double

__global__ void cudaMandelbrot2D(real X0, real Y0, real X1, real Y1, int POZ, int PION, int ITER,int *Mandel ){
    int k;
    real X_re, X_im, Y_re, Y_im, d_re, d_im, t_re;
    int inx = blockIdx.x * blockDim.x + threadIdx.x;
    int iny = blockIdx.y * blockDim.y + threadIdx.y;
    // int stepx = gridDim.x * blockDim.x;
    // int stepy = gridDim.y * blockDim.y;
    d_re = (X1-X0)/(POZ-1);
    d_im = (Y1-Y0)/(PION-1);
    if(inx<PION && iny<POZ){
                X_re= X0+ d_re*iny;
                X_im = Y0+ inx*d_im;
                Y_re= X_re;
                Y_im= X_im;
                k=0;
                while(k<ITER & (Y_re*Y_re + Y_im*Y_im)<4){
                    t_re = Y_re*Y_re-Y_im*Y_im+X_re;
                    Y_im = 2*Y_re*Y_im+X_im;
                    Y_re = t_re;
                    k++;
                }
                Mandel[inx*POZ+iny]= k;

    }
    
}

void swap(double *p,double *q) {
   double t;
   t=*p; 
   *p=*q; 
   *q=t;
}

double srednia(int len, double *tab){
    double sum=0;
    for(int i =1;i<len; i++ ){
        sum+=tab[i]/(len-1);
    }
    return sum;
}

double min(int len, double *tab){
    double sum=tab[0];
    for(int i =1;i<len; i++ ){
        if(tab[i]<sum)
            sum=tab[i];
    }
    return sum;
}

void computeMandelbrot(real X0, real Y0, real X1, real Y1, int POZ, int PION, int ITER,int *Mandel ){
    int k;
    real X_re, X_im, Y_re, Y_im, d_re, d_im, t_re;

    d_re = (X1-X0)/(POZ-1);
    d_im = (Y1-Y0)/(PION-1);

    for(int i =0; i<PION; i++){
        for(int j =0; j<POZ; j++){
            X_re= X0+ d_re*j;
            X_im = Y0+ i*d_im;
            Y_re= X_re;
            Y_im= X_im;
            k=0;
            while(k<ITER & (Y_re*Y_re + Y_im*Y_im)<4){
                t_re = Y_re*Y_re-Y_im*Y_im+X_re;
                Y_im = 2*Y_re*Y_im+X_im;
                Y_re = t_re;
                k++;
            }
            Mandel[i*POZ+j]= k;
        }
    }
}

int main(int argc, char **argv) {
    //Ustaw obszar obliczeń {X0,Y0} - lewy dolny róg
    double X0=-1.0;    double Y0=0.0;
    //{X1,Y1} - prawy górny róg
	  double X1=-0.5;    double Y1=0.5;
    //Ustal rozmiar w pikselach {POZ,PION}
    int POZ=1000; int PION=1000;
    //Ustal liczbę iteracji próbkowania {ITER}
    int ITER=256;

    //Zaalokuj tablicę do przechowywania wyniku
    int *Iters = (int*) malloc(sizeof(int)*POZ*PION);

    int* Iters_gpu;

    hipError_t status;

    status = hipMalloc((void**)&Iters_gpu, sizeof(int)* POZ*PION);
    if(status!= hipSuccess){ cout << hipGetErrorString(status) << endl;}
    
    //printf("Computations for rectangle { (%lf %lf), (%lf %lf) }\n",X0,Y0,X1,Y1);
    int block_szer;
    int block_wys;
    int ile_iter=50;
    float dt_ms;
    double wyniki[ile_iter];
    cout << "2D" << endl;
    for(int i=8; i<=32;i*=2){
        block_szer = i;
        block_wys = i;

        dim3 threadPerBlock(block_szer,block_wys,1);
        dim3 numBlocks(POZ/block_szer+1,PION/block_wys+1,1);
        for(int ij=0;ij<ile_iter;ij++)
        {
            auto start2 = chrono::steady_clock::now(); 
            cudaMandelbrot2D<<<numBlocks,threadPerBlock,0>>>(X0,Y0,X1,Y1,POZ,PION,ITER,Iters_gpu);
            status = hipMemcpy(Iters,Iters_gpu,sizeof(int)*POZ*PION, hipMemcpyDeviceToHost);
            if(status != hipSuccess){ cout << hipGetErrorString(status) << endl;}

            auto stop = chrono::steady_clock::now();
            auto diff = stop - start2;

            //cout << chrono::duration <double, milli> (diff).count() << endl;
            wyniki[ij]=chrono::duration <double, milli> (diff).count();
        }
        cout << block_szer << "x" << block_wys << " Średnia: " << srednia(ile_iter, wyniki) << " Min: "<< min(ile_iter, wyniki)<< endl;
    }
    
    status = hipFree(Iters_gpu);
    if(status != hipSuccess){ cout << hipGetErrorString(status) << endl;}
    int *Iters_cpu = (int*) malloc(sizeof(int)*POZ*PION);
    auto start2 = chrono::steady_clock::now();
    computeMandelbrot(X0,Y0,X1,Y1,POZ,PION,ITER,Iters_cpu);
     
    auto stop = chrono::steady_clock::now();
    auto diff = stop - start2;
    cout << "CPU czas :"<<chrono::duration <double, milli> (diff).count() << endl;
    cout << "Dla wymiaru obrazka:"<< POZ << "x" << PION <<endl;
   
    free(Iters);
}
