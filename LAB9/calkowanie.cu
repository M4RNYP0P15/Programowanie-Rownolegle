
#include <hip/hip_runtime.h>
#include <iomanip>
#include <iostream>
#include <cstdlib>
#include <chrono>

using namespace std;
//  mialem liczyc na karcie graficznej (device) ale wyniki się różniły o mw 0,003 - 0,008
template <typename T,unsigned int blockSize>
__device__ void warpReduce(volatile T *sdata, unsigned int tid) {
    if (blockSize >= 64) sdata[tid] += sdata[tid + 32];
    if (blockSize >= 32) sdata[tid] += sdata[tid + 16];
    if (blockSize >= 16) sdata[tid] += sdata[tid + 8];
    if (blockSize >= 8) sdata[tid] += sdata[tid + 4];
    if (blockSize >= 4) sdata[tid] += sdata[tid + 2];
    if (blockSize >= 2) sdata[tid] += sdata[tid + 1];
}
template <typename T,unsigned int blockSize>
__global__ void reduce6(T *g_idata, T *g_odata, unsigned int n) {
    extern __shared__ T sdata[];
    unsigned int tid = threadIdx.x;
    unsigned int i = blockIdx.x*(blockSize*2) + tid;
    unsigned int gridSize = blockSize*2*gridDim.x;
    sdata[tid] = 0;
    while (i < n) { 
        sdata[tid] += g_idata[i] + g_idata[i+blockSize]; 
        i += gridSize; 
    }
    
    __syncthreads();
    //printf("gridDim: %d i: %d tid[%d]: %lf\n",gridDim.x,i, tid, sdata[tid]);
    if (blockSize >= 512) { if (tid < 256) { sdata[tid] += sdata[tid + 256]; } __syncthreads(); }
    if (blockSize >= 256) { if (tid < 128) { sdata[tid] += sdata[tid + 128]; } __syncthreads(); }
    if (blockSize >= 128) { if (tid < 64) { sdata[tid] += sdata[tid + 64]; } __syncthreads(); }
    if (tid < 32) warpReduce<T,blockSize>(sdata, tid);
    if (tid == 0) g_odata[blockIdx.x] = sdata[0];
}

__device__ double f(double x) { return(x * x + 2 * x); }

__global__ void licz(long long N, double xp, double dx, double * pTab){
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    if(tid<=N) pTab[tid]=f(xp + tid * dx)*dx;
}

int main()
{
  int N = 3000; //liczba punktów/prostokątów podziałowych / liczba watkow
  double xp,xk,dx;
  double * pGPU;
//   double * pOGPU;

  cout << setprecision(3) << fixed;   // 3 cyfry po przecinku ; format stałoprzecinkowy
  cout << "Obliczanie  calki oznaczonej za pomoca metody prostokatow\n"
          "f(x) = x * x + 2 * x\n"
          "Podaj poczatek przedzialu calkowania\n"
          "xp = 1";
  //cin >> xp;
  xp=1.0;
  cout << "\nPodaj koniec przedzialu calkowania\n"
          "xk = 2";
  //cin >> xk;
  xk=2.0;
  cout << "\nPodaj dokladnosci calkowania\n"
          "n = ";
    cin >> N;
  cout << endl;


  dx = (xk - xp) / N;
  int threads = 128;
  long blocks = (N+threads-1)/threads;
//   int Ni = ((N+threads-1)/threads)/2;
    double sumaCPU=0;

  double *Iters = (double*) malloc(sizeof(double)*N+1);
//   double *Iters_cop = (double*) malloc(sizeof(double)*Ni);

  hipMalloc((void**)&pGPU, sizeof(double)* N+1);
//   cudaMalloc((void**)&pOGPU, sizeof(double)* Ni);
    auto start2 = chrono::steady_clock::now();
  licz<<<blocks, threads>>>(N, xp, dx, pGPU);
  hipMemcpy(Iters, pGPU, N* sizeof(double), hipMemcpyDeviceToHost);

    // dim3 dimBlock(threads, 1, 1);
    // dim3 dimGrid(blocks/2, 1, 1);

    // when there is only one warp per block, we need to allocate two warps
    // worth of shared memory so that we don't index shared memory out of bounds
    // int smemSize = (threads <= 32) ? 2 * threads * sizeof(double) : threads * sizeof(double);
    // switch (threads) {
    //     case 512:
    //       reduce5<512> <<<dimGrid, dimBlock, smemSize>>>(pGPU, pOGPU, N);
    //       break;

    //     case 256:
    //       reduce5<256>
    //           <<<dimGrid, dimBlock, smemSize>>>(pGPU, pOGPU, N);
    //       break;

    //     case 128:
        //   reduce6<double, 128> <<<dimGrid, dimBlock, smemSize>>>(pGPU, pOGPU, N);
    //       break;

    //     case 64:
    //       reduce5<64>
    //           <<<dimGrid, dimBlock, smemSize>>>(pGPU, pOGPU, N);
    //       break;

    //     case 32:
    //       reduce5<32>
    //           <<<dimGrid, dimBlock, smemSize>>>(pGPU, pOGPU, N);
    //       break;

    //     case 16:
    //       reduce5<16>
    //           <<<dimGrid, dimBlock, smemSize>>>(pGPU, pOGPU, N);
    //       break;

    //     case 8:
    //       reduce5<8>
    //           <<<dimGrid, dimBlock, smemSize>>>(pGPU, pOGPU, N);
    //       break;

    //     case 4:
    //       reduce5<4>
    //           <<<dimGrid, dimBlock, smemSize>>>(pGPU, pOGPU, N);
    //       break;

    //     case 2:
    //       reduce5<2>
    //           <<<dimGrid, dimBlock, smemSize>>>(pGPU, pOGPU, N);
    //       break;

    //     case 1:
    //       reduce5<1>
    //           <<<dimGrid, dimBlock, smemSize>>>(pGPU, pOGPU, N);
    //       break;
    //   }
//   cudaDeviceSynchronize();
//   cudaMemcpy(Iters_cop, pOGPU, Ni* sizeof(double), cudaMemcpyDeviceToHost);

//   cudaMemcpy (&s, pOGPU, 1 * sizeof(double), cudaMemcpyDeviceToHost);
  
    for(int h=1;h<=N;h++) sumaCPU+=Iters[h];

    auto stop = chrono::steady_clock::now();
  auto diff = stop - start2;
    cout << chrono::duration <double, milli> (diff).count() << endl;
    printf("\nSuma GPU: %lf\n", sumaCPU);
//   cout << "Wartosc calki wynosi : " << setw(8) << s << endl;
    // double suma=0;
    // for(int h=0;h<Ni;h++){
    //     suma+=Iters_cop[h];
    //     printf("[%lf]\n", Iters_cop[h]);
    // }
    // printf("%f\n", suma);
  return 0;
}


